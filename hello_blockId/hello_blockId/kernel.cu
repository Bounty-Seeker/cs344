
#include "hip/hip_runtime.h"


#include <stdio.h>

#define NUM_BLOCKS 32
#define BLOCK_WIDTH 1

__global__ void hello()
{
	printf("Hello world! I'm a thread in block %d\n", blockIdx.x);
}


int main(int argc, char **argv)
{
	// launch the kernel
	hello <<<NUM_BLOCKS, BLOCK_WIDTH >>>();

	// force the printf()s to flush
	hipDeviceSynchronize();

	printf("That's all!\n");

	// cudaDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	hipDeviceReset();
	
	getchar();

	return 0;
}